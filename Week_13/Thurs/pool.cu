#include "hip/hip_runtime.h"
#include <stdio.h>
#include <demo_util.h>

#define CLOCK_RATE 1080000     // in kHZ

__device__ uint get_smid(void) {

     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}

__device__ void sleep(float t)
{
    clock_t t0 = clock64();
    clock_t t1 = t0;
    while ((t1 - t0)/(CLOCK_RATE*1000.0f) < t)
    {
        t1 = clock64();
    }
}

__global__ void worker(float *t,int *s) 
{
    int id = blockIdx.x;
    s[id] = get_smid();
    sleep(t[id]);
}

#define N 64
#define MP 24

int main(void) 
{
    float *dev_t;
    int *dev_s;
    float t[N];
    int s[N];
    float SM[MP];
    int i;

    /* Allocate memory on the device */
    hipMalloc( (void**)&dev_t, N*sizeof(float));
    hipMalloc( (void**)&dev_s, N*sizeof(int));

    for(i = 0; i < MP; i++)
        SM[i] = 0;

    /* sleep time */
    for(i = 0; i < N; i++)
        t[i] = 1;

    hipMemcpy(dev_t, t, N*sizeof(float), hipMemcpyHostToDevice);

    dim3 block(N);
    dim3 grid(N);
    worker<<<grid,block>>>(dev_t, dev_s);

    /* Copy contents of dev_t back to t */
    hipMemcpy(s, dev_s, N*sizeof(float), hipMemcpyDeviceToHost);

    for(i = 0; i < N; i++)
    {
        printf( "Block %2d worked for %8.4f seconds on SM %d\n",i,t[i],s[i]);
        SM[s[i]] += t[i];
    }
    printf("\n");
    for(i = 0; i < MP; i++)
    {
        printf("SM[%2d] = %6.1f\n",i,SM[i]);
    }
    hipFree(dev_t);

}


