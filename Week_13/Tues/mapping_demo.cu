#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <cuda_util.h>
#include <demo_util.h>

__global__ void addmat(int m, int n, int* A, int *B, int *C) 
{
    /* Each thread processes one element */
    unsigned int ix = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y*blockDim.y;
    int idx = iy*m + ix; 
    if (ix < m && iy < n)   
        C[idx] = A[idx] + B[idx];
}


void addmat_host(int m, int n, int* A, int *B, int *C)
{
    int ix,iy,idx;
    for(iy = 0; iy < n; iy++)
        for(ix = 0; ix < m; ix++)
        {
            idx = iy*m + ix;
            C[idx] = A[idx] + B[idx];
        }
}


double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return (double) tp.tv_sec + (double)tp.tv_usec*1e-6;
}


int main(int argc, char** argv) 
{
    /* Host */
    int *A, *B, *C;

    /* Device */
    int *dev_A, *dev_B, *dev_C;

    /* scalars */
    size_t m, n, nbytes;
    double etime, start;

    int run_host;
    int dimx, dimy;

    int err0;
    read_int(argc, argv, "--host", &run_host, &err0);
    if (err0 > 0)
        run_host = 0;

    int err1, err2; 
    read_int(argc, argv, "--dimx", &dimx, &err1);
    read_int(argc, argv, "--dimy", &dimy, &err2);    

    if (err1 > 0 || err2 > 0)
    {
        printf("Problem reading dimx or dimy\n");
        exit(0);
    }

    /* Matrix is m x n */
    m = 1 << 14;  
    n = 1 << 14;  

    nbytes = m*n*sizeof(int);

    printf("--------------- Memory -------------\n");
    printf("%20s %10d\n","m",m);
    printf("%20s %10d\n","n",n);
    printf("%20s %10.1f (mb)\n","Memory",3*nbytes/(1024.0*1024.0));
    printf("%20s %10.1f (gb)\n","Memory",3*nbytes/(1024.0*1024.0*1024.0));
    printf("\n");

    A = (int*) malloc(nbytes);
    B = (int*) malloc(nbytes);
    C = (int*) malloc(nbytes);

    int k;
    for(k = 0; k < m*n; k++)
    {
        A[k] = 1;
        B[k] = 2;
    }

    /* Allocate memory on the device */
    printf("----------- Initialization ---------\n");
    start = cpuSecond();
    hipMalloc((void**) &dev_A, nbytes);
    hipMalloc((void**) &dev_B, nbytes);
    hipMalloc((void**) &dev_C, nbytes);
    etime = cpuSecond() - start;
    printf("%20s %10.3g (s)\n","hipMalloc",etime);

    start = cpuSecond();
    hipMemcpy(dev_A, A, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, nbytes, hipMemcpyHostToDevice);
    etime = cpuSecond() - start;
    printf("%20s %10.3g (s)\n","Copy (HtoD)",etime);


    /* Launch nx blocks, each with ny threads */

    dim3 block(dimx, dimy);  /* Distribute each row to blocks */
    dim3 grid((m+block.x-1)/block.x,(n+block.y-1)/block.y);

    start = cpuSecond();
    addmat<<<grid,block>>>(m,n,dev_A, dev_B, dev_C);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    etime = cpuSecond() - start;
    printf("%20s %10.3g (s)\n","GPU Kernel", etime);

    /* Copy contents from device back to host */
    start = cpuSecond();
    hipMemcpy(C, dev_C, nbytes, hipMemcpyDeviceToHost);
    etime = cpuSecond() - start;
    printf("%20s %10.3g (s)\n","Copy (DtoH)", etime);
    printf("\n");

    printf("---------------- Host --------------\n");
    if (run_host)
    {
        start = cpuSecond();
        addmat_host(m,n,A,B,C);
        etime = cpuSecond() - start;
        printf("%20s %10.3g (s)\n","Host",etime);        
    }
    else
    {
        printf("%20s %10s\n","Host","N/A");
    }

    printf("\n");
    printf("----------- Grid/Block Info ---------\n");
    printf("%20s %6d %6d","grid.x, block.x",grid.x, block.x);
    if (m <= block.x*grid.x)
    {
        printf("        m <= block.x*grid.x = %d\n",block.x*grid.x);
    }
    else
    {
        printf("        m > block.x*grid.x = %d\n",block.x*grid.x); 
    }
    printf("%20s %6d %6d","grid.y, block.y",grid.y, block.y);
    if (n <= block.y*grid.y)
    {
        printf("        m <= block.y*grid.y = %d\n",block.y*grid.y);
    }
    else
    {
        printf("        m > block.y*grid.y = %d\n",block.y*grid.y); 
    }
    if (dimx*dimy > 1024)
    {
        printf("\n");
        printf("-----> WARNING : Threads per block exceed 1024.\n");
        printf("\n");
    }



    printf("\n");

#if 0
    printf("\n");


    printf("\n");
    int i,j;
    k = 0;
    for(j = 0; j < n; j++)
    {
        for(i = 0; i < m; i++)
        {
            printf("C[%2d,%2d] = %d\n",i,j,C[k]);
            k++;
        }
        printf("\n");
    }
#endif    

    hipFree(dev_A);
    hipFree(dev_B);
    free(A);
    free(B);

    hipDeviceSynchronize();
    hipDeviceReset();
}


